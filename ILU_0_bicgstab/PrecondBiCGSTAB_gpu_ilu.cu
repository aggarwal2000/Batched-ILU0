#include "hip/hip_runtime.h"
#include<iostream>
#include<stdio.h>
#include<vector>
#include<cassert>
#include<chrono>
#include<cmath>
#include "hip/hip_runtime_api.h"
#include "matrix.h"
#include "ReadWriteData.h"
#include "header.h"
#include "PrecondBiCGSTAB.h"
#include "SolverResults.h"

#include "parILU_0.h"
#include "ILU_0.h"


//TODO: Move kernels like: Norm, Innerproduct, SpMV, sparse trsv to some other file.
//TODO: parallel reductions(norm, inner product)

namespace {

__device__ void ComputeResidualVec(const int num_rows,const int* const A_row_ptrs_shared,const int* const A_col_inds_shared,
    const double* const A_vals_shared,const double* const b_shared,const double* const x_shared, double* const res_shared)
{
    
    int num_warps_in_block = blockDim.x/WARP_SIZE;
    int lane = threadIdx.x & (WARP_SIZE -1);
    int local_warp_index = threadIdx.x/WARP_SIZE; //local warp index in a block

    for(int i = local_warp_index; i < num_rows ; i = i + num_warps_in_block)
    {
        int start_ind_for_row = A_row_ptrs_shared[i];
        int end_ind_for_row = A_row_ptrs_shared[i + 1];

        double temp = 0;

        for(int k = start_ind_for_row + lane; k < end_ind_for_row; k = k + WARP_SIZE)
        {
            temp += A_vals_shared[k]*x_shared[A_col_inds_shared[k]];
        }

        double val = temp;

        //warp level reduction
        for (int offset = 16; offset > 0; offset /= 2)
        val += __shfl_down_sync(FULL_MASK, val, offset);
    
        if(lane == 0)
        {   
            res_shared[i] = b_shared[i] - val;
        }

       

    }

}



__device__ void block_reduce(double* data)
{
    int nt = blockDim.x;
    int tid = threadIdx.x;

    for (int k = nt / 2; k > 0; k = k / 2)
    {
        __syncthreads();
        if (tid < k)
        {
            data[tid] += data[tid + k];
        }
    }


}

__device__ double inner_product1(const int num_rows, const double* const vec1_shared, const double* const vec2_shared, double* const temp_shared)
{   
    double tmp = 0;

    for(int i = threadIdx.x; i < num_rows; i = i + blockDim.x)
    {
        tmp = tmp + vec1_shared[i]*vec2_shared[i];
    }

    temp_shared[threadIdx.x] = tmp;

    __syncthreads();

    block_reduce(temp_shared);

    __syncthreads();

    return temp_shared[0];

}

__device__ double inner_product(const int num_rows, const double* const vec1_shared, const double* const vec2_shared)
{   
    double tmp = 0;

    for(int i=0; i < num_rows; i++)
        tmp = tmp + vec1_shared[i]*vec2_shared[i];
    

    __syncthreads();

   
   return tmp;

}



__device__ double L2Norm(const int num_rows,const double* const vec_shared)
{
    return sqrt(inner_product(num_rows,vec_shared,vec_shared));
}




__device__ void initialization(const int num_rows, const int num_nz,const int* const row_ptrs,const int* const col_inds,
    const double* const vals_mat,const double* const vals_rhs ,double* const  x_shared,double* const v_shared,double* const p_shared,
double* const r_shared,double* const r_hat_shared)
{
    int num_warps_in_block = blockDim.x/WARP_SIZE;
    int local_thread_id = threadIdx.x; //local thread id in block
    int local_warp_index = threadIdx.x/WARP_SIZE; //local warp index in a block
    int page_id = blockIdx.x;
    int lane  = threadIdx.x & (WARP_SIZE -1);

    
    // x:initialize with 0s {Later on, have a provision for user's choice. So, may be x_pages: initialize--> with something n copy that to here}
    // r = b - A*x
    // r_hat = r
    // rho, alpha, omega
    // v with 0s
    // p with 0s

   

    for(int i = local_thread_id ; i < num_rows; i = i + blockDim.x)
    {   
        x_shared[i] = 0.00;
        v_shared[i] = 0.00;
        p_shared[i] = 0.00;
        
    }

    __syncthreads();

    //initialize r
    ComputeResidualVec(num_rows, row_ptrs, col_inds, vals_mat + page_id*num_nz, vals_rhs + page_id*num_rows, x_shared,r_shared);
    __syncthreads();

    
    for(int i = local_warp_index*WARP_SIZE  + lane ; i < num_rows ; i = i + num_warps_in_block*WARP_SIZE)
    {   
        r_hat_shared[i] = r_shared[i];
    }

    
}


__device__ void Update_p(const int num_rows,double* const p_shared,const double* const r_shared,const double* const v_shared,
    const double beta,const double omega_old)
{
    
    for(int i = threadIdx.x ; i < num_rows; i = i + blockDim.x)
    {   
        double val = r_shared[i] + beta*(p_shared[i] - omega_old*v_shared[i]);
        p_shared[i] = val;
        
    }

} 


__device__ void Update_s(const int num_rows,double* const s_shared,const double* const r_shared,const double alpha,const double* const v_shared)
{
    for(int i = threadIdx.x; i < num_rows; i = i + blockDim.x)
    {
        s_shared[i] = r_shared[i] - alpha*v_shared[i];
    }
}

__device__ void Update_x(const int num_rows,double* const x_shared,const double* const p_shared,const double* const s_shared,const double alpha,
    const double omega_new)
{
    for(int i = threadIdx.x; i < num_rows; i = i + blockDim.x)
    {
        x_shared[i] = x_shared[i] + alpha*p_shared[i] + omega_new*s_shared[i];
    }
}


__device__ void Update_x_middle(const int num_rows, double* const x_shared,const double* const p_shared, const double alpha)
{   
    for(int i = threadIdx.x; i < num_rows; i = i + blockDim.x)
    {
        x_shared[i] = x_shared[i] + alpha*p_shared[i] ;
    }

}



__device__ void Update_r(const int num_rows,double* const r_shared,const double* const s_shared,const double* const t_shared,const double omega_new)
{
    
    for(int i = threadIdx.x; i < num_rows; i = i + blockDim.x)
    {
        r_shared[i] = s_shared[i] - omega_new*t_shared[i];
    }

    
}





__device__ void SpMV(const int num_rows,const int* const mat_row_ptrs_shared,const int* const mat_col_inds_shared,
    const double* const mat_vals_shared,const double* const vec_shared,double* const ans_shared)
{
  
    int num_warps_in_block = blockDim.x/WARP_SIZE;
    int lane = threadIdx.x & (WARP_SIZE -1);
    int local_warp_index = threadIdx.x/WARP_SIZE; //local warp index in a block

    for(int i = local_warp_index; i < num_rows ; i = i + num_warps_in_block)
    {
        int start_ind_for_row = mat_row_ptrs_shared[i];
        int end_ind_for_row = mat_row_ptrs_shared[i + 1];

        double temp = 0;

        for(int k = start_ind_for_row + lane; k < end_ind_for_row; k = k + WARP_SIZE)
        {
            temp += mat_vals_shared[k]*vec_shared[mat_col_inds_shared[k]];
        }

        double val = temp;

        //warp level reduction
        for (int offset = 16; offset > 0; offset /= 2)
        val += __shfl_down_sync(FULL_MASK, val, offset);
    

        if(lane == 0)
        {   
            ans_shared[i] = val;
        }

        

    }

}







__global__ void KernelFillTrueResNorms(const int num_rows, const int num_nz, const int num_pages, const int* const row_ptrs, 
    const int* const col_inds, const double* const vals_mat, const double* const vals_rhs, const double* const vals_ans, double* const true_residual_norms)
{
    __shared__ int A_row_ptrs_shared[MAX_NUM_ROWS + 1];
    __shared__ int A_col_inds_shared[MAX_NUM_NZ];
    __shared__ double A_vals_shared[MAX_NUM_NZ];
    __shared__ double b_shared[MAX_NUM_ROWS];
    __shared__ double x_shared[MAX_NUM_ROWS];

    __shared__ double r_true_shared[MAX_NUM_ROWS];

    int page_id = blockIdx.x;
    
    if(page_id < num_pages)
    {

            for(int i = threadIdx.x; i < num_rows + 1; i = i + blockDim.x)
            {   
                A_row_ptrs_shared[i] = row_ptrs[i];

            }


            for(int i = threadIdx.x ; i < num_nz; i = i + blockDim.x)
            {   
                A_col_inds_shared[i] = col_inds[i];
                A_vals_shared[i] = vals_mat[i + page_id*num_nz];

            }

            for(int i = threadIdx.x ; i < num_rows; i = i + blockDim.x)
            {   
                b_shared[i] = vals_rhs[i + page_id*num_rows];
                x_shared[i] = vals_ans[i + page_id*num_rows];   
            }

            __syncthreads();

            ComputeResidualVec(num_rows, A_row_ptrs_shared, A_col_inds_shared, A_vals_shared, b_shared, x_shared,r_true_shared);
            __syncthreads();


            double true_resi_norm = L2Norm(num_rows,r_true_shared);

            if(threadIdx.x == 0)
                true_residual_norms[page_id] = true_resi_norm;
    }



}    


__device__ void legacy_sparse_lower_triangular_solve(const int num_rows, const int* const L_row_ptrs, const int* const L_col_idxs, 
const double* const L_values, const double* const vec_shared, volatile double* const temp_vec_shared)
{   
   
        const int row_index = threadIdx.x;

        if(row_index >= num_rows)
        {
            return;
        }

        double sum = 0;

        const int start = L_row_ptrs[row_index];
        const int end = L_row_ptrs[row_index + 1] - 1;
        int i = start;
        
        
        bool completed = false;

        while(!completed)
        {   
            

            const int col_index = L_col_idxs[i];

            if( i < end  &&  isfinite(temp_vec_shared[col_index]))
            {
                sum += L_values[i] * temp_vec_shared[col_index];
                i++;
            }

           
            if(i == end)
            {   
                temp_vec_shared[row_index] = (vec_shared[row_index] - sum)/L_values[end];
               
                completed = true;
               
            }

          
        }

       
        
}


__device__ void legacy_sparse_upper_triangular_solve(const int num_rows,  const int* const U_row_ptrs, const int* const U_col_idxs, 
const double* const U_values, volatile const double* const temp_vec_shared, volatile double* const vec_hat_shared)
{
    const int row_index = threadIdx.x;

    if(row_index >= num_rows)
    {
        return;
    }

    double sum = 0;

    const int start = U_row_ptrs[row_index];
    const int end = U_row_ptrs[row_index + 1]  - 1;
    int i = end;

    bool completed = false;

    while(!completed )
    {   
       

        const int col_index = U_col_idxs[i];

        if( i > start && isfinite(vec_hat_shared[col_index]))
        {
            sum += U_values[i] * vec_hat_shared[col_index];
            i--;
        }

      
        if(i == start)
        {
            vec_hat_shared[row_index] = (temp_vec_shared[row_index] - sum)/U_values[start];
           
            completed = true;
        }

      
    }

}

__device__ void ApplyPreconditionerILU(const int num_rows , const int* const L_row_ptrs, 
    const int* const L_col_idxs , const double* const L_values,  const int* const U_row_ptrs,
    const int* const U_col_idxs, const double* const U_values,  const double* const vec_shared,volatile double* const vec_hat_shared)
{

   
    // vec_hat = precond * vec
    // => L * U  * vec_hat = vec
    // => L * y = vec , find y , and then U * vec_hat = y, find vec_hat

    // we need sparse triangular solves for that!
    //if we want to use the busy waiting while loop approach, then the num_rows should be <= threadblock size, else there is possibility of a deadlock !

    //TODO: For upper trsv, use thread 0 for the bottommost row, this way we could avoid :  assert(num_rows <= blockDim.x), as there won't be a possibility of deadlock then!

    
    assert(num_rows <= blockDim.x);

    __shared__  volatile double temp_vec_shared[MAX_NUM_ROWS];

    for(int i = threadIdx.x ; i < num_rows; i += blockDim.x)
    {
        temp_vec_shared[i] = 1.8/0; //TODO: find a better way to deal with this!
        vec_hat_shared[i] = 1.3/0;

    }

    __syncthreads();
    
    

    legacy_sparse_lower_triangular_solve(num_rows,  L_row_ptrs, L_col_idxs, L_values, vec_shared, temp_vec_shared);

    __syncthreads();

    

    legacy_sparse_upper_triangular_solve(num_rows,  U_row_ptrs, U_col_idxs, U_values, temp_vec_shared, vec_hat_shared);

    

}




__global__ void KernelBatchedPreconditionedBiCGSTAB(const int num_rows, const int num_nz, const int num_pages, const int* const row_ptrs, 
    const int* const col_inds, const double* const vals_mat, const double* const vals_rhs, double* const vals_ans,
    const int L_nnz , const int* const L_row_ptrs, const int* const L_col_idxs, const double* const L_vals ,
    const int U_nnz, const int* const U_row_ptrs, const int* const U_col_idxs, const double* const U_vals,
    float* const iter_counts , int* const conv_flags, double* const iter_residual_norms)
{
  /* ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~shared memory ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ */

   

    //This won't work for isoocatne
    //Except for isooctane, the nrows is atmost 54, and nnz: 2560 ... 
    
    //TODO: Use dynamic shared memory
    /*
        --> Now it is easy to use dynamic shared memory as we don't need to store ints !
        --> But with isooctane, there is no warning/error about the shared memory limits; kernel is simply not launched (if dynamic shared mem greater than what is available is used), this leads to wrong results! Pending: Check with cuda get last error...
    */

    // __shared__ int row_ptrs_shared[MAX_NUM_ROWS + 1];
    // __shared__ int col_idxs_shared[MAX_NUM_NZ];
    // __shared double values_shared[MAX_NUM_NZ];

    __shared__ double x_shared[MAX_NUM_ROWS];
    __shared__ double r_shared[MAX_NUM_ROWS];
    __shared__ double r_hat_shared[MAX_NUM_ROWS];
    __shared__ double p_shared[MAX_NUM_ROWS];
    __shared__ double v_shared[MAX_NUM_ROWS];
    __shared__ double s_shared[MAX_NUM_ROWS];
    __shared__ double t_shared[MAX_NUM_ROWS];
   // __shared__ double r_true_shared[MAX_NUM_ROWS];
    

    __shared__ double s_hat_shared[MAX_NUM_ROWS];
    __shared__ double p_hat_shared[MAX_NUM_ROWS];
 


    int page_id = blockIdx.x;


    if(page_id < num_pages)
    {   


        
        /*~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ shared memory initialization/assigments~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ */
        initialization(num_rows, num_nz, row_ptrs, col_inds, vals_mat, vals_rhs, x_shared, v_shared, p_shared, r_shared, r_hat_shared);
        
        __syncthreads();


        /*--------------------------------------------------- Preconditioner already generated ----------------------------------------------------*/


        double res_initial = L2Norm(num_rows, r_shared); 
          
        double iter_residual_norm = res_initial;

        double rho_old = 1;
        double rho_new = 1;
        double omega_old = 1;
        double omega_new = 1;
        double alpha = 1;
        double beta = 1; 

        double b_norm = L2Norm(num_rows, vals_rhs + page_id*num_rows);
        
        int conv_flag = -1;



        if(b_norm == 0)
        {   
            for(int i = threadIdx.x; i < num_rows ; i += blockDim.x)
                x_shared[i] = 0;


            if(threadIdx.x == 0 )
            {   
                printf(" RHS for problem id: %d is 0. x = 0 is the solution. ",page_id);

                iter_counts[page_id] = 0;
                conv_flags[page_id] = 1;
                iter_residual_norms[page_id] = 0;
            }    

            __syncthreads();
        
        }
        else
        {
            if(res_initial < ATOL )
            {   
                if(threadIdx.x == 0 )
                {   
                    printf("\n Initial guess for problem id: %d is good enough. No need of iterations. \n", page_id);


                    iter_counts[page_id] = 0;
                    conv_flags[page_id] = 1;
                    iter_residual_norms[page_id] = res_initial;
                }	    
            }
            else
            {
                 /*~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ Then can start iterating ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ */
                    float iter = 0;
                
                    while(iter < MAX_ITER)
                    {
                        rho_new = inner_product(num_rows, r_shared, r_hat_shared);

                        if(rho_new == 0)
                        {
                            if(threadIdx.x == 0)
                            {
                                printf("\n Method failed for problem id: %d\n",page_id);
                            }

                            break;
                        }
                        
                        beta = (rho_new/rho_old)*(alpha/omega_old);
                    
                        
                        Update_p(num_rows,p_shared,r_shared ,v_shared,beta,omega_old);
                        __syncthreads();
                        

                        ApplyPreconditionerILU(num_rows, L_row_ptrs, L_col_idxs, L_vals + page_id * L_nnz, U_row_ptrs, U_col_idxs , U_vals + page_id * U_nnz, p_shared, p_hat_shared);

                        __syncthreads();

                        SpMV(num_rows, row_ptrs,col_inds, vals_mat + page_id*num_nz, p_hat_shared, v_shared);
                        __syncthreads(); 
                        
                        
                        double r_hat_and_v_inner_prod = inner_product(num_rows,r_hat_shared,v_shared);
                        alpha = rho_new/r_hat_and_v_inner_prod;        
                    

                        Update_s(num_rows,s_shared,r_shared,alpha,v_shared);
                        __syncthreads();
                        

                        iter_residual_norm = L2Norm(num_rows, s_shared); //an estimate
                        
                        iter = iter + 0.5;

                        if( iter_residual_norm < ATOL)
                        {
                            Update_x_middle(num_rows,x_shared,p_hat_shared,alpha);
                            __syncthreads();

                            conv_flag = 1;

                            
                            break;
    
                        }

                     
                        ApplyPreconditionerILU(num_rows, L_row_ptrs, L_col_idxs, L_vals + page_id * L_nnz, U_row_ptrs, U_col_idxs , U_vals + page_id * U_nnz, s_shared, s_hat_shared);
                        __syncthreads();


                        SpMV( num_rows, row_ptrs , col_inds, vals_mat + page_id*num_nz , s_hat_shared, t_shared);
                        __syncthreads();
                    


                        double t_and_s_inner_prod = inner_product(num_rows,t_shared,s_shared);
                        double t_and_t_inner_prod = inner_product(num_rows,t_shared,t_shared);
                        omega_new = t_and_s_inner_prod/t_and_t_inner_prod;
                        

                        Update_x(num_rows,x_shared,p_hat_shared,s_hat_shared,alpha,omega_new);
                        __syncthreads();
                        
                        
                        iter = iter + 0.5;


                        Update_r(num_rows,r_shared,s_shared,t_shared,omega_new);
                        __syncthreads();

                        iter_residual_norm = L2Norm(num_rows,r_shared);
                        rho_old = rho_new;
                        omega_old = omega_new;

                        if( iter_residual_norm < ATOL)
                        {   
                            conv_flag = 1;
                            break;
                        }

                        
                    }

                    __syncthreads();

                   /*  ComputeResidualVec(num_rows, row_ptrs , col_inds, vals_mat + page_id*num_nz, vals_rhs + page_id*num_rows, x_shared,r_true_shared);
                    __syncthreads();
                
                    
                    double true_resi_norm = L2Norm(num_rows,r_true_shared); */
                    
                    if(threadIdx.x == 0 )
                    {   
                      // printf("\nConv flag for problem_id: %d is %d , iter resi norm : %0.17lg, true resi norm: %0.17lg, iter:%f ",page_id,conv_flag, iter_residual_norm, true_resi_norm, iter );
                        iter_counts[page_id] = iter;
                        conv_flags[page_id] = conv_flag;
                        iter_residual_norms[page_id] = iter_residual_norm;
                    }

            }

        }

       
       // At the end,copy x_shared to global memory.
        for(int i = threadIdx.x; i < num_rows; i += blockDim.x)
            vals_ans[i + page_id*num_rows] = x_shared[i];

    
    }

}




int Batched_BiCGSTAB_Gpu_helper(const PagedCSRMatrices & A_pages,const PagedVectors& b_pages,PagedVectors & x_pages, SolverResults & solver_results,const bool is_parilu , const int num_iter_par_ilu )
{
    std::cout << "\n\n-------------------------------------------------------------------------------\n Batched_Preconditioned BiCGSTAB_Gpu_helper " << std::endl;
    
   
    auto start = std::chrono::high_resolution_clock::now();
    
     //generate ILU preconditioner
    PagedCSRMatrices L_pages;
    PagedCSRMatrices U_pages;


    if(is_parilu)
    {	
    	//std::cout << " \npar ilu with num iter: " << num_iter_par_ilu << std::endl; 
        ParILU_0_Factorization_Gpu(A_pages , L_pages, U_pages, num_iter_par_ilu);
    }
    else
    {
        const int approach_num = 1;
        //Note: For pele matrices, approach 1 works better as compared to the depenedency graph approach as the matrices are not that sparse. For other cases, approach 3 is exepected to be faster than others.
        //std::cout << " \nilu " << std::endl;
        ILU_0_Factorization_Gpu(A_pages , L_pages, U_pages, approach_num);
    }
    	
	
    dim3 block(THREADS_PER_BLOCK,1,1);
    dim3 grid_solver(A_pages.GetNumPages(),1,1 );

    //------------------------------------------------------------------------------- Call main solver kernel-------------------------------------------------//

    KernelBatchedPreconditionedBiCGSTAB<<< grid_solver, block , 0  >>>(A_pages.GetNumRows(), A_pages.GetNumNz(), A_pages.GetNumPages(),
    A_pages.GetPtrToGpuRowPtrs(),A_pages.GetPtrToGpuColInd(), A_pages.GetPtrToGpuValues(), b_pages.GetPtrToGpuValues(), x_pages.GetPtrToGpuValues(),
    L_pages.GetNumNz(), L_pages.GetPtrToGpuRowPtrs(), L_pages.GetPtrToGpuColInd(), L_pages.GetPtrToGpuValues(), 
    U_pages.GetNumNz() ,U_pages.GetPtrToGpuRowPtrs(), U_pages.GetPtrToGpuColInd(), U_pages.GetPtrToGpuValues(),
    solver_results.GetPtrToGpuIterCount(), solver_results.GetPtrToGpuConvFlag() , solver_results.GetPtrToGpuIterResNorm());

    hipDeviceSynchronize();

    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);

    

    std::cout << "\n Batched Preconditioned BiCGSTAB on GPU is done!!!" << std::endl;
    std::cout << " Time taken is: "  << (double)duration.count() << " microseconds\n\n ";  

    solver_results.SetTimeTaken((double)duration.count()/ 1000);


    //fill it with true residual norms
    KernelFillTrueResNorms<<< grid_solver , block , 0  >>>(A_pages.GetNumRows(), A_pages.GetNumNz(), A_pages.GetNumPages(), A_pages.GetPtrToGpuRowPtrs(),
    A_pages.GetPtrToGpuColInd(), A_pages.GetPtrToGpuValues(), b_pages.GetPtrToGpuValues(), x_pages.GetPtrToGpuValues(), solver_results.GetPtrToGpuTrueResNorm());

    hipDeviceSynchronize();

    return 1;
}



} //unnamed namespace


//----------------------------------------------------------------------------------------------------------------------------------------------------------------






// A*x = b
void Batched_ILU_Preconditioned_BiCGSTAB_Gpu(const std::vector<std::string> & subdir, const PagedCSRMatrices & A_pages,const PagedVectors& b_pages,PagedVectors & x_pages,const bool is_scaled,  SolverResults & solver_results , const bool is_parilu , const int num_iter_par_ilu  )
{
    assert(A_pages.ExistsGPU() == true);
    assert(b_pages.ExistsGPU() == true);
    assert(x_pages.ExistsGPU() == true);

    const int num_pages = A_pages.GetNumPages();
    assert(num_pages == b_pages.GetNumPages());
    assert(num_pages == x_pages.GetNumPages());

    const int num_rows = A_pages.GetNumRows();
    const int num_cols = A_pages.GetNumCols();
    
    assert(num_rows == num_cols);
    assert(num_cols == x_pages.GetNumElements());
    assert(num_rows == b_pages.GetNumElements());

   
    int success_code = 0;


    success_code = Batched_BiCGSTAB_Gpu_helper(A_pages,b_pages,x_pages, solver_results, is_parilu, num_iter_par_ilu);

    std::string solution_file;

    if(is_scaled == true)
        solution_file = "x_scaled_gpu_ilu_bicgstab.mtx";
    else
        solution_file = "x_gpu_ilu_bicgstab.mtx";

    if(success_code == 1)
    {
        x_pages.CopyFromGpuToCpu();
        Print_ans(subdir,x_pages, solution_file);
        std::cout << "files containing soluation: x  are produced...  ( " <<  solution_file <<  " ) in their respective directories " << std::endl;

    }


}
